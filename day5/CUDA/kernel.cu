﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cinttypes>
#include <array>
#include <vector>
#include <string>
#include <chrono>
#include <sstream>
#include <fstream>

#define MAX_RANGES_PER_MAP 64

#define TIMESTAMP(name) const auto name = std::chrono::high_resolution_clock::now();

struct MapRange {
    uint32_t src_start{}, src_end{}, dst_start{};
};
struct SeedRange {
    uint32_t start{}, len{};
};

int read_file(std::vector<SeedRange>& seeds, std::vector<std::array<MapRange, 64>>& ranges);

__device__ uint32_t map_src_to_dst(uint32_t src, MapRange ranges[], uint32_t range_idx) {
    MapRange* range = &ranges[range_idx * MAX_RANGES_PER_MAP];
    
    for (uint32_t i = 0U; range[i].src_end != 0; ++i) {
        if (src < range[i].src_start) {
            continue;
        }
    
        if (src > range[i].src_end) {
            continue;
        }
    
        return range[i].dst_start + src - range[i].src_start;
    }

    return src;
}

__global__ void iter_ranges_map(uint32_t* min, SeedRange* seed, MapRange ranges[]) {
    uint32_t index = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (index >= seed->len)
        return;

    uint32_t location = 
    map_src_to_dst(
        map_src_to_dst(
            map_src_to_dst(
                map_src_to_dst(
                    map_src_to_dst(
                        map_src_to_dst(
                            map_src_to_dst(
                                seed->start + index, ranges, 0
                            ), ranges, 1
                        ), ranges, 2
                    ), ranges, 3
                ), ranges, 4
            ), ranges, 5
        ), ranges, 6
    );

    if (location < *min) {
        *min = location;
    }
}

void get_min_for_seed_range(uint32_t* minimum, SeedRange* seed, MapRange ranges[], uint32_t seed_range_len) {
    dim3 threadsPerBlock(8U, 4U); // SM is 32 = 8 x 4
    dim3 blockCount(((seed_range_len - 1U) / (threadsPerBlock.x * threadsPerBlock.y)) + 1U);

    iter_ranges_map<<<blockCount, threadsPerBlock>>>(minimum, seed, ranges);
}

int main() {
    std::vector<SeedRange> seeds{};
    std::vector<std::array<MapRange, MAX_RANGES_PER_MAP>> ranges{}; // End MapRange is filled with zeros 

    if (read_file(seeds, ranges)) {
        printf("failed to read file!\n");
        return 1;
    }

    SeedRange* seeds_device{};
    MapRange* ranges_device{};

    hipMalloc(&seeds_device, sizeof(SeedRange) * seeds.size());
    hipMemcpy(seeds_device, seeds.data(), sizeof(SeedRange) * seeds.size(), hipMemcpyHostToDevice);

    hipMalloc(&ranges_device, sizeof(MapRange) * MAX_RANGES_PER_MAP * ranges.size());
    hipMemcpy(ranges_device, ranges.data(), sizeof(MapRange) * MAX_RANGES_PER_MAP * ranges.size(), hipMemcpyHostToDevice);

    uint32_t minimum = UINT32_MAX;
    uint32_t* minimum_device{};

    hipMalloc(&minimum_device, sizeof(uint32_t));
    hipMemcpy(minimum_device, &minimum, sizeof(uint32_t), hipMemcpyHostToDevice);

TIMESTAMP(start)

    for (uint32_t i{}; i < seeds.size(); ++i) {
        get_min_for_seed_range(minimum_device, &seeds_device[i], ranges_device, seeds[i].len);
    }

    hipMemcpy(&minimum, minimum_device, sizeof(uint32_t), hipMemcpyDeviceToHost);

TIMESTAMP(end)

    printf("Time: %.02fs\n", std::chrono::duration<float>(end - start).count());
    printf("Result: %u\n", minimum);

    hipFree(seeds_device);
    hipFree(ranges_device);
    hipFree(minimum_device);
}

int read_file(std::vector<SeedRange>& seeds, std::vector<std::array<MapRange, 64>>& ranges) {
    std::ifstream file("input.txt");

    if (!file.is_open()) {
        printf("failed to open file!\n");
        return 1;
    }

    std::string line{};
    std::getline(file, line); // seeds

    line = line.substr(6);
    std::stringstream ss(line);

    uint32_t n0, n1;
    while ((ss >> n0) && (ss >> n1)) {
        seeds.push_back(SeedRange{ n0, n1 });
    }

    std::getline(file, line); // blank space

    ranges.resize(7);

    // 7 Maps
    for (int i{}; i < 7; ++i) {
        std::getline(file, line); // name

        //printf("%d:\n", i);

        uint32_t j{};
        while (true) {
            if (!std::getline(file, line)) break;

            if (line == "") break;

            uint32_t a, b, c;
            if (!sscanf(line.c_str(), "%u %u %u", &a, &b, &c)) {
                printf("sscanf fail!\n");
                return 1;
            }

            ranges[i][j++] = MapRange{ b, b + c - 1, a };
        }
    }

    return 0;
}